#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <iostream>
//#include <immintrin.h>
#include <hip/hip_runtime.h>

#define N 32 // matrix size


// function to print a matrix
void print_matrix(int *matrix, int rows, int cols) {
  for (int i = 0; i < rows; i++) {
    for (int j = 0; j < cols; j++) {
      printf("%d ", matrix[i * cols + j]);
    }
    printf("\n");
  }
}


__global__ void calcC(int * A, int * B, int *C,int NN){
int i=blockIdx.x*blockDim.x+threadIdx.x;
int j=blockIdx.y*blockDim.y+threadIdx.y;
C[i*NN+j]=0;
for (int k=0; k<NN;k++){
C[i*NN+j]+=A[i*NN+k]*B[k*NN+j];
}
}

int main(int argc, char **argv) {
  int *A, *B, *C; // matrices

  // allocate memory for matrices
   A = (int *)malloc(N * N * sizeof(int));
   B = (int *)malloc(N * N * sizeof(int));
   C = (int *)malloc(N * N * sizeof(int));
int *Adev,*Bdev,*Cdev;

hipMalloc(&Adev,N*N*sizeof(int));
hipMalloc(&Bdev,N*N*sizeof(int));
hipMalloc(&Cdev,N*N*sizeof(int));


  // initialize matrices with random values
    srand(0); // seed for reproducibility
    for (int i = 0; i < N * N; i++) {
      A[i] = rand() % 10;
      B[i] = 0;//rand() % 10;
    }
for (int i=0;i<N;i++){B[i*N+i]=1;}
    printf("Matrix A:\n");
    print_matrix(A, N, N);
    printf("Matrix B:\n");
    print_matrix(B, N, N);
   const auto start_chrono = std::chrono::high_resolution_clock::now();
hipMemcpy(Adev,A,N*N*sizeof(int),hipMemcpyHostToDevice);
hipMemcpy(Bdev,B,N*N*sizeof(int),hipMemcpyHostToDevice);
  // perform matrix multiplication on local matrices
  //for (int i=0; i<N;i++ ){
    //for (int j=0; j<N; j++){
      //C[i*N+j]=0;

    //   for (int k=0; k<N; k++){
    //     C[i*N+j]+=A[i*N+k]*B[k*N+j];
    //   }

    //}
  //}
dim3 numBlocks(N,N,1);
dim3 numThreads(1,1,1);

calcC<<<numBlocks,numThreads>>>(Adev,Bdev,Cdev,N);
hipMemcpy(C,Cdev,N*N*sizeof(int),hipMemcpyDeviceToHost);
    const auto finish_chrono = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> diff = finish_chrono - start_chrono;

   //print the result matrix
    printf("Matrix C:\n");
    print_matrix(C, N, N);


  // free memory and finalize MPI
  free(A);
  free(B);
  free(C);

hipFree(Adev);
hipFree(Bdev);
hipFree(Cdev);

   std::cout << "Runtime: " << diff.count() << " s. " << std::endl;

  return 0;
}
