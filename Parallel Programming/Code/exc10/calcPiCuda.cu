#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <cmath>
#include <hip/hip_runtime.h>



#define PI 3.14159265358979323846  /* pi */

double integral(double boundaryLeft, double boundaryRight, long divisions);
double f(double x);

__global__ void integralCuda(double *res,double global_a, double global_b,long divisions, long number_of_processes){

    double tmp = 0.0;
    int th =threadIdx.x;
          //  long local_number_of_intervals = divisions / number_of_processes;
        double local_interval_width = (global_b - global_a) / (double)number_of_processes;

        double local_a = global_a + th * local_interval_width;
        double local_b = local_a + local_interval_width;

    double locIntervalWidth = (local_b - local_a) / (double)divisions;
    for (int i = 0; i < divisions; i++)
    {
        tmp += fdev(local_a + locIntervalWidth * ((double)i + 0.5)) * locIntervalWidth;
    }
    res[th]=tmp;

}

__device__ double fdev(double x)
{
    return (std::sqrt(1. - x * x));
}



int main(int argc, char **argv)
{
    int number_of_processes = 30;
    int my_rank = 0;
    const int repetitions = 1;

    double result = 0.0;
    double local_integral = 0.0;
    const auto start_chrono = std::chrono::high_resolution_clock::now();

    const long number_of_intervals = 5040 * 100000;

    const double global_a = 0.0;
    const double global_b = 1.0;
    double *res,*resHost;
    int numThreads=128;
    hipMalloc(&res,numThreads*sizeof(double));
    resHost=(double*)malloc(numThreads*sizeof(double));
    for (int round = 0; round < repetitions; round++)
    {
    //     long local_number_of_intervals = number_of_intervals / number_of_processes;
    //     double local_interval_width = (global_b - global_a) / (double)number_of_processes;
    //    // #pragma omp parallel for default(shared) private(my_rank,local_integral)
    //     for (my_rank=0;my_rank<number_of_processes;my_rank++){

    //     double local_a = global_a + my_rank * local_interval_width;
    //     double local_b = local_a + local_interval_width;

    //     local_integral = integral(local_a, local_b, local_number_of_intervals);

        // send / receive
        
       //  result += local_integral;
       integralCuda<<<1,numThreads>>>(res,global_a,global_b,number_of_intervals,numThreads);
       hipMemcpy(resHost,res,numThreads*sizeof(double), hipMemcpyDeviceToHost);
        for (int i=0; i<numThreads;i++){result+=resHost[i];}

                 }
       // my_rank=0;

    }

    result = result / repetitions;

    const auto finish_chrono = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> diff = finish_chrono - start_chrono;
    if (my_rank == 0)
    {
        std::cout << "Runtime: " << diff.count() << " s. " << std::endl;
        printf("Result: %0.20f \n", result * 4.);
        printf("Error: %0.20f \n", fabs(result * 4 - PI));
    }

    hipFree(res);
    free(resHost);
    return 0;
}

double integral(double boundaryLeft, double boundaryRight, long divisions)
{
    double tmp = 0.0;
    double locIntervalWidth = (boundaryRight - boundaryLeft) / (double)divisions;
    //#pragma omp parallel for default(shared) private(tmp)
    for (int i = 0; i < divisions; i++)
    {
        tmp += f(boundaryLeft + locIntervalWidth * ((double)i + 0.5)) * locIntervalWidth;
    }
    return tmp;
}

double f(double x)
{
    return (std::sqrt(1. - x * x));
}
