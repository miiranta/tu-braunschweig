#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <iostream>
#include <immintrin.h>

#include <hip/hip_runtime.h>

#define N 16 // matrix size


// function to print a matrix
void print_matrix(int *matrix, int rows, int cols) {
  for (int i = 0; i < rows; i++) {
    for (int j = 0; j < cols; j++) {
      printf("%d ", matrix[i * cols + j]);
    }
    printf("\n");
  }
}

__global__ void matrixMultiplicationKernel(int* A, int* B, int* C)
{

  int ROW = blockIdx.y*blockDim.y+threadIdx.y;
  int COL = blockIdx.x*blockDim.x+threadIdx.x;

  float tmpSum = 0;

  if (ROW < N && COL < N) {
      // each thread computes one element of the block sub-matrix
      for (int i = 0; i < N; i++) {
          tmpSum += A[ROW * N + i] * B[i * N + COL];
      }
  }
  C[ROW * N + COL] = tmpSum;
}

int main(int argc, char **argv) {
  int *A, *B, *C; // matrices

  // allocate memory for matrices
   A = (int *)malloc(N * N * sizeof(int));
   B = (int *)malloc(N * N * sizeof(int));
   C = (int *)malloc(N * N * sizeof(int));

  // initialize matrices with random values
    srand(0); // seed for reproducibility
    for (int i = 0; i < N * N; i++) {
      A[i] = rand() % 10;
      B[i] = rand() % 10;
    }
    printf("Matrix A:\n");
    print_matrix(A, N, N);
    printf("Matrix B:\n");
    print_matrix(B, N, N);
   const auto start_chrono = std::chrono::high_resolution_clock::now();

  // perform matrix multiplication on local matrices

  int THR_PER_BLOCK = 32;
  int THR_BLOCK_SIZE = N/THR_PER_BLOCK;

  dim3 blocksPerGrid(THR_BLOCK_SIZE, 1, 1);
  dim3 threadsPerBlock(THR_PER_BLOCK, 1, 1);

  matrixMultiplicationKernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C);


  const auto finish_chrono = std::chrono::high_resolution_clock::now();
  std::chrono::duration<float> diff = finish_chrono - start_chrono;

  //print the result matrix
  printf("Matrix C:\n");
  print_matrix(C, N, N);


  // free memory and finalize MPI
  free(A);
  free(B);
  free(C);

   std::cout << "Runtime: " << diff.count() << " s. " << std::endl;

  return 0;
}

